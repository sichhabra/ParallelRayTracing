#include "hip/hip_runtime.h"
#include "imager.h"
#include "antialias.h"

namespace Imager{
#if __CUDA_ARCH__ < 600
    __device__ double atomicAdd(double* address, double val)
    {
        unsigned long long int* address_as_ull =
            (unsigned long long int*)address;
        unsigned long long int old = *address_as_ull, assumed;

        do {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val +
                        __longlong_as_double(assumed)));

            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
        } while (assumed != old);

        return __longlong_as_double(old);
    }
#endif 
    __global__ void _anti_alias(double *addr_r,double *addr_g,double *addr_b,double **red,double **green,double **blue,int i,int j,int antiAliasFactor){
        int di=threadIdx.x;
        int dj=threadIdx.y;
        int x=antiAliasFactor*i + di;
        int y=antiAliasFactor*j + dj;
        atomicAdd(addr_r,red[x][y]);
        atomicAdd(addr_g,green[x][y]);
        atomicAdd(addr_b,blue[x][y]);
    }

    Color cuda_antiAlias(double **red,double **green,double **blue,int i,int j,int antiAliasFactor,int wide,int height){

        double *addr_r,*addr_g,*addr_b,*rr,*gg,*bb;
        double **r,**g,**b;
        hipMallocManaged(&addr_r,4);
        hipMallocManaged(&addr_g,4);
        hipMallocManaged(&addr_b,4);
        hipMalloc(&r,wide*height*sizeof(double));
        hipMalloc(&g,wide*height*sizeof(double));
        hipMalloc(&b,wide*height*sizeof(double));
        hipMemset(addr_r,0,sizeof(double));
        hipMemset(addr_g,0,sizeof(double));
        hipMemset(addr_b,0,sizeof(double));

        hipMemcpy(r,red,wide*height*sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(g,green,wide*height*sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(b,blue,wide*height*sizeof(double),hipMemcpyHostToDevice);

        _anti_alias<<<antiAliasFactor,antiAliasFactor>>>(addr_r,addr_g,addr_b,red,green,blue,i,j,antiAliasFactor);

	rr=(double*)malloc(sizeof(double));
	gg=(double*)malloc(sizeof(double));
	bb=(double*)malloc(sizeof(double));

	hipMemcpy(rr,addr_r,sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(gg,addr_g,sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(bb,addr_b,sizeof(double),hipMemcpyDeviceToHost);
        
        return Color(*rr,*gg,*bb);
    }
}
