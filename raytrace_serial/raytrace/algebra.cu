#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include "algebra.h"
namespace Algebra{

//const double TOLERANCE = 1.0e-8;

__global__ void filter_real(int N,const std::complex<double> *inArray, double *outArray){

    int i = blockIdx.x;
    if (fabs(inArray[i].imag()) < TOLERANCE)
    {
        outArray[i] = inArray[i].real();
    }

}

int cuda_FilterRealNumbers(int numComplexValues, const std::complex<double> inArray[], double outArray[]){

    std::complex<double> *a;
    double *b;
    int N=numComplexValues;
    
    double temp[N];
    for(int i=0;i<N;i++) temp[i]=TOLERANCE;

    hipMalloc(&a,N*sizeof(std::complex<double>));
    hipMalloc(&b,N*sizeof(double));

    hipMemcpy(a, inArray, (N)*sizeof(std::complex<double>), hipMemcpyHostToDevice);
    filter_real<<<N,1>>>(N,a,b);
    
    hipMemcpy(temp, b, (N)*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(a);
    hipFree(b);

    int count=0;
    for(int i=0;i<N;i++){
        if(temp[i]!=TOLERANCE){
            outArray[count++]=temp[i];
        }
    }

    return count;
}
}
