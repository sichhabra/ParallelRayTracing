#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include "algebra.h"
namespace Algebra{

__global__ void filter_real(int N,const std::complex<double> *inArray, double *outArray){

    int i = blockIdx.x;
    if (fabs(inArray[i].imag()) < TOLERANCE)
    {
        outArray[i] = inArray[i].real();
    }

}

int cuda_FilterRealNumbers(int numComplexValues, const std::complex<double> inArray[], double outArray[]){

    std::complex<double> *a;
    double *b;
    int N=numComplexValues;
    
    hipMalloc(&a,N*sizeof(std::complex<double>));
    hipMalloc(&b,N*sizeof(double));

    hipMemcpy(a, inArray, (N)*sizeof(std::complex<double>), hipMemcpyHostToDevice);
    filter_real<<<N,1>>>(N,a,b);
    
    hipMemcpy(outArray, b, (N)*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(a);
    hipFree(b);

    return N;
}
}
